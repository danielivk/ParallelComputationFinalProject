#include "hip/hip_runtime.h"
#include "Header.h"
#include "hip/hip_runtime.h"

__device__ char compareCharacters(char c1, char c2);
__global__ void calculateSimilarityChar(char* seq1, char* seq2, char* signs, int length, int offset, int mutationIndex);
__constant__ char conservativeGroups[9][5] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
__constant__ char semiConservativeGroups[11][7] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};

 
__global__ void calculateSimilarityChar(char* seq1, char* seq2, char* signs, int length, int offset, int mutationIndex) 
{
//	Fill the signs array with a similarity sequence as this: ::*..*.**:: * . :: .

    	int i = blockDim.x * blockIdx.x + threadIdx.x;
	char c1, c2;
    	// compare the characters and write to signs in the corresponding positions accoarding to the thread ID 
    	if (i < length)
	{
		c1 = seq1[i + offset];
		c2 = seq2[i];
		if(i == mutationIndex)
		{
			*(signs + i) = ' ';
			return;
		}
		else if(i > mutationIndex)
			c2 = seq2[i - 1];

		*(signs + i ) = compareCharacters(c1, c2);
	}

}


__device__ char compareCharacters(char c1, char c2)
{	
//	Given two Characters, compute the result Character with regard to the conservative and semi-conservative groups.
	char* s;
	int i, j, containsC1 = 0, containsC2 = 0;
	char c;
	
	
	if(c1 == c2)
		return '*';
	for(i=0; i<9 ;i++)
	{
		s = conservativeGroups[i];
		containsC1 = 0;
		containsC2 = 0;
		for (j=0; j<4; j++) 
		{
			c = s[j];
			if(c == '\0')
				break;
			if(c == c1)
			{
				containsC1 = 1;
				if(containsC2)
					return ':';
			}
			if(c == c2)
			{
				containsC2 = 1;
				if(containsC1)
					return ':';
			}				
		}
		
	}
	
	for(i=0; i<11 ;i++)
	{
		s = semiConservativeGroups[i];		
		containsC1 = 0;
		containsC2 = 0;	
		for (j=0; j<6; j++) 
		{
			c = s[j];
			if(c == '\0')
				break;
			if(c == c1)
			{
				containsC1 = 1;
				if(containsC2)
					return '.';
			}
			if(c == c2)
			{
				containsC2 = 1;
				if(containsC1)
					return '.';
			}				
		}

	}
	return ' ';

}
		

hipError_t allocateCudaMemory(char** seq1, char** seq2, char** signs, int msLength, int sLength)
{
	// Given String array pointers, allocate them into the CUDA memory.

	// Error code to check return values for CUDA calls
    	hipError_t err1 = hipSuccess;
    	hipError_t err2 = hipSuccess;
    	hipError_t err3 = hipSuccess;


    	size_t size1 = msLength * sizeof(char);
    	size_t size2 = sLength * sizeof(char);
	
	

	
    	// Allocate memory on GPU to copy the data from the host
    	err1 = hipMalloc(seq1, size1);
    	err2 = hipMalloc(seq2, size2);
    	err3 = hipMalloc(signs, size2);
	
	if (err1 != hipSuccess) 
	{
        	fprintf(stderr, "1Failed to allocate device memory - %s\n", hipGetErrorString(err1));
        	exit(EXIT_FAILURE);
    	}

	if (err2 != hipSuccess) 
	{
        	fprintf(stderr, "2Failed to allocate device memory - %s\n", hipGetErrorString(err2));
        	exit(EXIT_FAILURE);
    	}
	
	if (err3 != hipSuccess) 
	{
        	fprintf(stderr, "3Failed to allocate device memory - %s\n", hipGetErrorString(err3));
        	exit(EXIT_FAILURE);
    	}

	return hipSuccess;

}

hipError_t copyInformationToCuda(char* seq1, char* seq2, MainSequence* ms, Sequence* s)
{
	// Given empty String pointers after alocation inside CUDA, copy the original information to them for further computation.
	// Error code to check return values for CUDA calls
    	hipError_t err1 = hipSuccess;
    	hipError_t err2 = hipSuccess;



    	size_t size1 = ms->length * sizeof(char);
    	size_t size2 = s->length * sizeof(char);
	
	
	// Copy data from host to the GPU memory
	err1 = hipMemcpy(seq1, ms->letters, size1, hipMemcpyHostToDevice);
	err2 = hipMemcpy(seq2, s->letters, size2, hipMemcpyHostToDevice);

	if (err1 != hipSuccess || err2 != hipSuccess) 
	{
		fprintf(stderr, "4Failed to copy data from host to device - %s\n", hipGetErrorString(err1));
		exit(EXIT_FAILURE);
	}
	
	if (err2 != hipSuccess) 
	{
		fprintf(stderr, "5Failed to copy data from host to device - %s\n", hipGetErrorString(err2));
		exit(EXIT_FAILURE);
	}

	return hipSuccess;

}

void freeCudaMemory(char* seq1, char* seq2, char* signs)
{

	// Free allocated memory on GPU
	if (hipFree(signs) != hipSuccess || hipFree(seq1) != hipSuccess || hipFree(seq2) != hipSuccess) 
	{
		fprintf(stderr, "8Failed to free device data");
		exit(EXIT_FAILURE);
	}

}

int GPU_Create_Signs(Sequence* s, int n, char* originalSigns, int mutationIndex, char* seq1, char* seq2, char* signs)
{
	// Given the Alocated and copied String Sequences, use the CUDA kernel to compute the target similarity String according to the given mutation and offset n.
	// Error code to check return values for CUDA calls
    	hipError_t err = hipSuccess;
    	size_t size = s->length * sizeof(char);
	
	
	// Launch the Kernel
	int threadsPerBlock = 100;
	int blocksPerGrid = (s->length + threadsPerBlock) / threadsPerBlock; // added + 1 for '-'

	calculateSimilarityChar<<<blocksPerGrid, threadsPerBlock>>>(seq1, seq2, signs, s->length + 1, n, mutationIndex); // including the '-' (this is the + 1)

	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		fprintf(stderr, "6Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(originalSigns, signs, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) 
	{
		fprintf(stderr, "7Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

		
	
	return 0;
}
